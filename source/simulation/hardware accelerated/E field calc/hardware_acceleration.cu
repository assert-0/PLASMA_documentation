#include "hip/hip_runtime.h"
#include <cstdio>
#include <windows.h>
#include <ctime>

#define DLL_BUILD

#include "hardware_acceleration.h"

#define PI 3.14159265357989323
#define MI 0.0000001
#define NTHREADS 1024
#define REDUCTIONFACTOR (NTHREADS * 2)
#define NREPEATS (150 * 150)
#define CHUNKSIZE (150 * 10)

#define TYPE float

#define FREQUENCY 1

#define INPUT 0
#define STATUS 0
#define DEBUG_COPPER_EXTENSION 0

long long freq;

const int nFi = 12;
const int nA = 20;
const int nB = 60;

struct param
{
	param(){ZeroMemory(this, sizeof(*this));}
	int nA;
	int nB;
	int nFi;
	TYPE j;
	TYPE Theta;
	TYPE IncFi;
	TYPE IncA;
	TYPE IncB;
	TYPE a;
	TYPE b;
	TYPE R;
	TYPE r;
	TYPE f;
	TYPE DWeights[4][4];
	TYPE FiWeights[::nFi + 1];
	TYPE EdgeList1[::nA + 1];
	TYPE EdgeList2[::nB + 1];
	TYPE *res;
};

__global__
void Calculate_e(param par)
{
	int index = threadIdx.x;
	int global_index = blockIdx.x * blockDim.x + index;
	
    TYPE G = 0, E = 0, F = 0, eR = 0, er = 0, C = 0;
	int ii = 0, ji = 0, ki = 0;
	TYPE Temp = 0;

    TYPE z = par.r * cos(par.Theta);
    TYPE q = par.r * sin(par.Theta);
    TYPE C0 = par.IncA * par.IncB * par.j * MI * par.IncFi;
    TYPE bh = par.b / 2;
	
	TYPE dR = 0;
	TYPE dr = -bh;
	TYPE dFi = 0;
	dR += TYPE(((global_index / (par.nFi + 1) / (par.nB + 1)) % (par.nA + 1)) * par.IncA);
	if(dR == 0 && global_index > (par.nFi + 1) * (par.nB + 1))
		return;
	dr += TYPE(((global_index / (par.nFi + 1)) % (par.nB + 1)) * par.IncB);
	dFi += TYPE((global_index % (par.nFi + 1)) * par.IncFi);
	
	eR = par.R + dR;
	ii = int(round(dR / par.IncA));
	C = C0 * eR;
	F = 2 * eR * q;
	er = z + dr;
	ji = int(round((dr + bh) / par.IncB));
	E = q * q + eR * eR + er * er;
	
	if(ii != par.nA && ii != 0 && ji != par.nB && ji != 0)
		Temp = par.DWeights[ii % 4][ji % 4];
	else if(ii == par.nA || ii == 0)
		Temp = par.EdgeList2[ji];
	else
		Temp = par.EdgeList1[ii];
	
	ki = int(round(dFi / par.IncFi));
	G = cos(dFi);
	par.res[global_index] = Temp * par.FiWeights[ki] * C * (G / (sqrt(E - F * G)));
}

__device__ 
void Warp_reduce(volatile TYPE *ar, unsigned int idx) {
	if (NTHREADS >= 64) ar[idx] += ar[idx + 32];
	if (NTHREADS >= 32) ar[idx] += ar[idx + 16];
	if (NTHREADS >= 16) ar[idx] += ar[idx + 8];
	if (NTHREADS >= 8) ar[idx] += ar[idx + 4];
	if (NTHREADS >= 4) ar[idx] += ar[idx + 2];
	if (NTHREADS >= 2) ar[idx] += ar[idx + 1];
}

__global__ 
void Cut_buffer(unsigned int n, TYPE *dest_buff, TYPE *src_buff) {
	__shared__ TYPE ar[NTHREADS];
	unsigned int idx = threadIdx.x;
	unsigned int i = blockIdx.x * (NTHREADS * 2) + idx;
	unsigned int grid_size = NTHREADS * 2 * gridDim.x;
	ar[idx] = 0;
	if(i >= n) return;
	while (i < n) 
	{ 
		ar[idx] += src_buff[i];
		if(i + NTHREADS < n)
			ar[idx] += src_buff[i + NTHREADS]; 
		i += grid_size; 
	}
	__syncthreads();
	if (NTHREADS >= 1024) { if (idx < 512) { ar[idx] += ar[idx + 512]; } __syncthreads(); }
	if (NTHREADS >= 512) { if (idx < 256) { ar[idx] += ar[idx + 256]; } __syncthreads(); }
	if (NTHREADS >= 256) { if (idx < 128) { ar[idx] += ar[idx + 128]; } __syncthreads(); }
	if (NTHREADS >= 128) { if (idx < 64) { ar[idx] += ar[idx + 64]; } __syncthreads(); }
	if (idx < 32) Warp_reduce(ar, idx);
	if (idx == 0) dest_buff[blockIdx.x] = ar[0];
}

void Cut_multiple_buffers(unsigned int src_size, TYPE *dest_buff, TYPE *src_buff, unsigned int buffer_size)
{
	int cycles = src_size / buffer_size;
	for(int a = 0; a < cycles; a++)
	{
		Cut_buffer<<<buffer_size / NTHREADS / 2 + 1, NTHREADS>>>(buffer_size,
																 dest_buff + a * int(buffer_size / NTHREADS / 2 + 1),
																 src_buff + a * buffer_size);
	}
	hipDeviceSynchronize();
}

void Calculate_single_point(param par, int blocks, TYPE *rese)
{
	if(par.r < 0.0)
	{
		hipMemset(rese, 0, blocks * NTHREADS * sizeof(TYPE));
		return;
	}
	
	par.res = rese;
	Calculate_e<<<blocks, NTHREADS>>>(par);
	
}
	
int last_alloc_size = 0;
TYPE *ine = nullptr, *result_transfer_buff = nullptr, *local_buff = nullptr;
TYPE DWeights[4][4];
TYPE FiWeights[nFi + 1];
TYPE EdgeList1[nA + 1];
TYPE EdgeList2[nB + 1];
TYPE FirstMultiplier = 506.25, SecondMultiplier = 11.25;
TYPE Moderator = FirstMultiplier * SecondMultiplier;

void Resource_cleanup()
{
	free(local_buff);
	hipFree(ine);
	hipFree(result_transfer_buff);
	local_buff = nullptr;
	ine = nullptr;
	result_transfer_buff = nullptr;
}

void Resource_startup(param *par, int num_ops)
{
	Resource_cleanup();
	int dimx, dimy, dimz, blocks;
	dimx = par->nA + 1;
	dimy = par->nB + 1;
	dimz = par->nFi + 1;
	blocks = (dimx * dimy * dimz) / NTHREADS + 1;
	hipMalloc(&ine, num_ops * sizeof(TYPE) * blocks * NTHREADS);
	hipMalloc(&result_transfer_buff, num_ops * sizeof(TYPE) * int(blocks * NTHREADS / REDUCTIONFACTOR + 1));
	local_buff = (TYPE*)malloc(num_ops * sizeof(TYPE) * int(blocks * NTHREADS / REDUCTIONFACTOR + 1));
	last_alloc_size = num_ops * sizeof(TYPE) * blocks * NTHREADS;
}

void Prepare_weights(param par)
{
	for(int k = 0; k <= nFi; k++)
	{
		if(k == nFi || k == 0)
			FiWeights[k] = 7;
		else if(k % 4 == 0)
			FiWeights[k] = 14;
		else if(k % 4 == 1 || k % 4 == 3)
			FiWeights[k] = 32;
		else if(k % 4 == 2)
			FiWeights[k] = 12;
	}
	for(int ii = 0; ii < 4; ii++)
	{
		for(int ij = 0; ij < 4; ij++)
		{
			if(ij % 4 == 0 && ii % 4 == 0)																		
				DWeights[ii][ij] = 196 / Moderator;
			else if(ii % 4 == 0 && ij % 4 == 1 || ii % 4 == 1 && ij % 4 == 0 || ii % 4 == 3 && ij % 4 == 0 || ii % 4 == 0 && ij % 4 == 3)	
				DWeights[ii][ij] = 448 / Moderator;
			else if(ii % 4 == 2 && ij % 4 == 0 || ii % 4 == 0 && ij % 4 == 2)
				DWeights[ii][ij] = 168 / Moderator;
			else if(ii % 4 == 1 && ij % 4 == 1 || ii % 4 == 3 && ij % 4 == 3 || ii % 4 == 3 && ij % 4 == 1 || ii % 4 == 1 && ij % 4 == 3)
				DWeights[ii][ij] = 1024 / Moderator;
			else if(ii % 4 == 2 && ij % 4 == 1 || ii % 4 == 1 && ij % 4 == 2 || ii % 4 == 3 && ij % 4 == 2 || ii % 4 == 2 && ij % 4 == 3)
				DWeights[ii][ij] = 384 / Moderator;
			else if(ij % 4 == 2 && ii % 4 == 2)
				DWeights[ii][ij] = 144 / Moderator;
		}
	}
	for(int m = 0; m <= nA; m++)
	{
		if(m == 0 || m == nA)
			EdgeList1[m] = 49 / Moderator;
		else if(m % 2 == 1)
			EdgeList1[m] = 224 / Moderator;
		else if(m % 4 == 0)		
			EdgeList1[m] = 98 / Moderator;
		else if(m % 4 == 2)
			EdgeList1[m] = 84 / Moderator;
	}
	for(int n = 0; n <= par.nB; n++)
	{
		if(n == 0 || n == par.nB)		
			EdgeList2[n] = 49 / Moderator;
		else if(n % 2 == 1)
			EdgeList2[n] = 224 / Moderator;
		else if(n % 4 == 0)
			EdgeList2[n] = 98 / Moderator;
		else if(n % 4 == 2)		
			EdgeList2[n] = 84 / Moderator;
	}
}

void Copy_ar(TYPE *ar1, TYPE *ar2, int size)
{
	memcpy(ar1, ar2, size * sizeof(TYPE));
}

void Calculate_hardware_accelerated(int num_ops, TYPE *theta, TYPE *point_distance, 
									TYPE j, TYPE inner_radius, TYPE length, 
									TYPE thickness, TYPE inc_thickness, TYPE inc_length, 
									TYPE inc_fi, TYPE *E, TYPE *deprecated_1, TYPE *deprecated_2)
{
	long long start, stop;
	
	int dimx, dimy, dimz, blocks;
	param par;
	par.nA = nA;
	par.nB = nB;
	par.nFi = nFi;
	par.j = j;
	par.IncFi = PI / nFi;
	par.IncA = thickness / nA;
	par.IncB = length / nB;
	par.a = thickness;
	par.b = length;
	par.R = inner_radius;
	par.f = FREQUENCY;
	
	dimx = par.nA + 1;
	dimy = par.nB + 1;
	dimz = par.nFi + 1;
	blocks = (dimx * dimy * dimz) / NTHREADS + 1;
	
	Prepare_weights(par);
	for(int it1 = 0; it1 < 4; it1++)
	{
		for(int it2 = 0; it2 < 4; it2++)
			par.DWeights[it1][it2] = DWeights[it1][it2];
	}
	Copy_ar(par.FiWeights, FiWeights, nFi + 1);
	Copy_ar(par.EdgeList1, EdgeList1, nA + 1);
	Copy_ar(par.EdgeList2, EdgeList2, nB + 1);
	
	if(last_alloc_size < num_ops * sizeof(TYPE) * blocks * NTHREADS)
	{
		Resource_startup(&par, num_ops);
	}
	
	#if DEBUG_COPPER_EXTENSION
		QueryPerformanceCounter((LARGE_INTEGER*)&start);
	#endif
	for(int a = 0; a < num_ops; a++)
	{
		par.Theta = theta[a];
		par.r = point_distance[a];
		Calculate_single_point(par, blocks, ine + a * blocks * NTHREADS);
	}
	hipDeviceSynchronize();
	#if DEBUG_COPPER_EXTENSION 
		QueryPerformanceCounter((LARGE_INTEGER*)&stop); 
		printf("	Calculations: %.15g s\n", double(stop - start) / double(freq));
	#endif
	
	
	
	#if DEBUG_COPPER_EXTENSION
		printf("	E\n");
	#endif
	if(E != nullptr)
	{
		#if DEBUG_COPPER_EXTENSION 
			QueryPerformanceCounter((LARGE_INTEGER*)&start);
		#endif
		Cut_multiple_buffers(num_ops * blocks * NTHREADS, result_transfer_buff, ine, blocks * NTHREADS);
		#if DEBUG_COPPER_EXTENSION 
			QueryPerformanceCounter((LARGE_INTEGER*)&stop);
			
			printf("		Buffer reduction: %.15g s\n", double(stop - start) / double(freq));
		
			QueryPerformanceCounter((LARGE_INTEGER*)&start);
		#endif
		hipMemcpy(local_buff, result_transfer_buff, num_ops * sizeof(TYPE) * int(blocks * NTHREADS / REDUCTIONFACTOR + 1), hipMemcpyDeviceToHost);
		for(int a = 0; a < num_ops; a++)
		{
			E[a] = 0;
			for(int b = 0; b < int(blocks * NTHREADS / REDUCTIONFACTOR + 1); b++)
			{
				E[a] += local_buff[a * int(blocks * NTHREADS / REDUCTIONFACTOR + 1) + b];
			}
			E[a] *= 2 * PI * par.f;
			deprecated_1[a] = 0;
			deprecated_2[a] = 0;
		}
		#if DEBUG_COPPER_EXTENSION 
			QueryPerformanceCounter((LARGE_INTEGER*)&stop);
			
			printf("		Data loading and processing: %.15g s\n", double(stop - start) / double(freq));
		#endif
	}



	#if DEBUG_COPPER_EXTENSION
		printf("	Buff1 size: %d\n", num_ops * int(sizeof(TYPE)) * blocks * NTHREADS);
		printf("	Buff2 size: %d\n", num_ops * int(sizeof(TYPE)) * int(blocks * NTHREADS / REDUCTIONFACTOR + 1));
		printf("	Reduction factor: %d\n", REDUCTIONFACTOR);
		printf("	Payload size: %d\n", num_ops * int(sizeof(TYPE)) * int(blocks * NTHREADS / REDUCTIONFACTOR + 1));
		printf("	Threads per calculation: %d\n", blocks * NTHREADS);
	#endif
}

TYPE j = 1000000;
TYPE inner_radius = 0.03;
TYPE thickness = 0.03;
TYPE length = 0.12;
TYPE alpha = PI / 6;

TYPE point_distance = 0.025;
TYPE theta = PI / 2;

TYPE inc_fi = PI / nFi;
TYPE inc_thickness = thickness / nA;
TYPE inc_length = length / nB;

TYPE *E, *filler;

TYPE *distance_ar, *theta_ar;

#if INPUT == 1

	void Input()
	{
		FILE *f = fopen("input.dat", "point_distance");
		fscanf(f, " j=%f", &j);
		fscanf(f, " point_distance=%f", &point_distance);
		fscanf(f, " inner_radius=%f", &inner_radius);
		fscanf(f, " thickness=%f", &thickness);
		fscanf(f, " length=%f", &length);
		fscanf(f, " theta=%f", &theta);
		fscanf(f, " alpha=%f", &alpha);
		fscanf(f, " inc_inner_radius=%f", &inc_thickness);
		fscanf(f, " inc_length=%f", &inc_length);
		fscanf(f, " inc_fi=%f", &inc_fi);
	}
	
#else

	void Input()
	{
		
	}

#endif

void Setup()
{
	E = new TYPE [NREPEATS];
	filler = new TYPE [NREPEATS];
	ZeroMemory(E, NREPEATS * sizeof(TYPE));
	distance_ar = new TYPE [NREPEATS];
	theta_ar = new TYPE [NREPEATS];
	for(int a = 0; a < NREPEATS; a++)
	{
		distance_ar[a] = point_distance;
		theta_ar[a] = theta;
	}
}

int main()
{
	Input();
	Setup();
	long long start, stop, start2, stop2;
	QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
	QueryPerformanceCounter((LARGE_INTEGER*)&start);
	for(int a = 0; a < NREPEATS / CHUNKSIZE; a++)
	{
		if(STATUS)
		{
			system("cls");
			printf("%.2f%% done...\n", (float)a / (NREPEATS / CHUNKSIZE) * 100.0);
		}
		#if DEBUG_COPPER_EXTENSION
			printf("Pass: %d\n", a);
			QueryPerformanceCounter((LARGE_INTEGER*)&start2);
		#endif
		Calculate_hardware_accelerated(CHUNKSIZE, theta_ar + a * CHUNKSIZE, 
									   distance_ar + a * CHUNKSIZE, j, inner_radius, 
									   length, thickness, inc_thickness, inc_length, 
									   inc_fi, E + a * CHUNKSIZE, filler + a * CHUNKSIZE, filler + a * CHUNKSIZE);
		#if DEBUG_COPPER_EXTENSION
			QueryPerformanceCounter((LARGE_INTEGER*)&stop2);
			printf("Elapsed time: %.15g s\n", double(stop2 - start2) / double(freq));
		#endif
	}
	QueryPerformanceCounter((LARGE_INTEGER*)&stop);
	
	Resource_cleanup();
	
	printf("Done!\n");
	printf("%.15f\n", E[NREPEATS - 1]);
	printf("Average time expended per operation: %.15g s\n", double(stop - start) / double(freq) / NREPEATS);
	printf("Total time expended: %.15g s\n", double(stop - start) / double(freq));
	system("pause");
	
	
	
//	--------------------------
	
	
	
	/*
	const int factor = NTHREADS * 2, buff1_size = 19200, buff2_size = buff1_size / factor == 0 ? 1 : buff1_size / factor + 1;
	printf("buff2 %d\n", buff2_size);
	TYPE res = 0;
	TYPE *loc1, *loc2, *dev1, *dev2;
	loc1 = (TYPE*)malloc(buff1_size * sizeof(TYPE));
	loc2 = (TYPE*)malloc(buff2_size * sizeof(TYPE));
	hipMalloc(&dev1, buff1_size * sizeof(TYPE));
	hipMalloc(&dev2, buff2_size * sizeof(TYPE));
	for(int a = 0; a < buff1_size; a++)
		loc1[a] = 1;
	ZeroMemory(loc2, buff2_size * sizeof(TYPE));
	hipMemcpy(dev1, loc1, buff1_size * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemset(dev2, 0, buff2_size * sizeof(TYPE));
	for(int a = 0; a < 25000; a++)
		Cut_buffer<<<buff2_size + 2, NTHREADS>>>(buff1_size, dev2, dev1);
	hipDeviceSynchronize();
	hipMemcpy(loc2, dev2, buff2_size * sizeof(TYPE), hipMemcpyDeviceToHost);
	for(int a = 0; a < buff2_size; a++)
		res += loc2[a];
	printf("%.15g\n", res);
	*/
	
	
	
	/*
	const unsigned int factor = NTHREADS * 2, chunk_size = 57600, buff1_size = chunk_size * CHUNKSIZE;
	const unsigned int buff2_size = (chunk_size / factor + 1) * CHUNKSIZE;
	long long start, stop;
	printf("%d\n", buff2_size);
	TYPE res = 0;
	TYPE *loc1, *loc2, *dev1, *dev2;
	QueryPerformanceFrequency((LARGE_INTEGER*)&freq);
	loc1 = (TYPE*)malloc(buff1_size * sizeof(TYPE));
	loc2 = (TYPE*)malloc(buff2_size * sizeof(TYPE));
	hipMalloc(&dev1, buff1_size * sizeof(TYPE));
	hipMalloc(&dev2, buff2_size * sizeof(TYPE));
	for(int a = 0; a < buff1_size; a++)
		loc1[a] = 1;
	ZeroMemory(loc2, buff2_size * sizeof(TYPE));
	hipMemcpy(dev1, loc1, buff1_size * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemset(dev2, 0, buff2_size * sizeof(TYPE));
	QueryPerformanceCounter((LARGE_INTEGER*)&start);
	Cut_multiple_buffers(buff1_size, dev2, dev1, chunk_size);
	hipMemcpy(loc2, dev2, buff2_size * sizeof(TYPE), hipMemcpyDeviceToHost);
	for(unsigned int a = 0; a < buff2_size; a++)
	{
		res += loc2[a];
	}
	res /= CHUNKSIZE;
	QueryPerformanceCounter((LARGE_INTEGER*)&stop);
	
	printf("res: %.15g\n", res);
	printf("%.15g s\n", double(stop - start) / double(freq));
	*/
	
	
	
	/*
	float tmp_ar[2048];
	ZeroMemory(tmp_ar, 2048 * sizeof(int));
	for(int a = 0; a < buff2_size; a++)
	{
		for(int b = 0; b < 2048; b += 2)
		{
			if(tmp_ar[b] == loc2[a])
			{
				tmp_ar[b + 1]++;
				break;
			}
			else if(tmp_ar[b] == 0)
			{
				tmp_ar[b] = loc2[a];
				tmp_ar[b + 1] = 1;
				break;
			}
		}
	}
	*/
	
	
	
	/*
	for(int b = 0; b < 2048; b += 2)
	{
		if(tmp_ar[b] == 0)
			break;
		printf("key: %f value: %f\n", tmp_ar[b], tmp_ar[b + 1]);
	}
	*/
	
	
	
	/*
	TYPE *loc1, *dev1, *dev2;
	TYPE res;
	int buff_size = 1027;
	
	loc1 = (TYPE*)malloc(buff_size * sizeof(TYPE));
	hipMalloc(&dev1, buff_size * sizeof(TYPE));
	hipMalloc(&dev2, buff_size * sizeof(TYPE));
	
	for(int a = 0; a < buff_size; a++)
		loc1[a] = 1;
	
	hipMemcpy(dev1, loc1, buff_size * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemcpy(dev2, loc1, buff_size * sizeof(TYPE), hipMemcpyHostToDevice);
	hipMemset(dev2, 0, buff_size * sizeof(TYPE));
	
	Reduce_buffer(buff_size, &res, dev1, dev2);
	
	printf("%.15g\n", res);
	*/
	
	
	
	/*
	hipFree(dev1);
	hipFree(dev2);
	free(loc1);
	free(loc2);
	*/
}